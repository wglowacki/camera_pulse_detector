#include "hip/hip_runtime.h"
#include "../include/matrixoperations.h"
#include <opencv2/opencv.hpp>
#include <thrust/extrema.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include <math.h>

#define N 10

namespace matrix_operations {

__global__ void multiply(int n, double *x, double *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] * y[i];
}

std::vector<double>
multiplyVecKernel( const std::vector<double>& first,
                   const std::vector<double>& second )
{
    double *x = nullptr, *y = nullptr;
    int arraySize = first.size();

    hipMallocManaged(&x, arraySize*sizeof(double));
    hipMallocManaged(&y, arraySize*sizeof(double));
    hipDeviceSynchronize();

    hipMemcpy(x, &first.front(),
               arraySize*sizeof(double),
               hipMemcpyHostToDevice
    );
    hipMemcpy(y, &second.front(),
               arraySize*sizeof(double),
               hipMemcpyHostToDevice
    );
    hipDeviceSynchronize();

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    multiply<<<numBlocks, blockSize>>>(arraySize, x, y);
    hipDeviceSynchronize();
    std::vector<double> retVect(arraySize);
    hipMemcpy(&retVect.front(), y,
               arraySize*sizeof(double),
               hipMemcpyDeviceToHost
    );
    hipDeviceSynchronize();
    // Free memory
    hipFree(x);
    hipFree(y);
    hipDeviceSynchronize();

    return retVect;
}

std::vector<double> calcLinspaceTimes (
        int vectSize, double startT, double endT )
{
    std::vector<double> hostEvenTimes(vectSize);
    thrust::device_vector<double> gpuEvenTimes = hostEvenTimes;
//        std::cout << endT << " " << startT << std::endl;
    double timeGap = (endT - startT) / (vectSize - 1);
    assert(timeGap>0);

    thrust::sequence(gpuEvenTimes.begin(), gpuEvenTimes.end(),
                     hostEvenTimes.front(), timeGap);
    thrust::transform(thrust::device,
                      gpuEvenTimes.begin(), gpuEvenTimes.end(),
                      thrust::make_constant_iterator(startT),
                      gpuEvenTimes.begin(),
                      thrust::plus<double>()
    );

    gpuEvenTimes.back() = endT;
    thrust::copy(gpuEvenTimes.begin(),
                 gpuEvenTimes.end(),
                 hostEvenTimes.begin()
    );
    hipDeviceSynchronize();
    return (hostEvenTimes);
}

std::vector<double> normalize(const std::vector<double>& vect)
{
    thrust::device_vector<float> dVect = vect;
    double norm = (thrust::reduce(dVect.begin(), dVect.end(), 0))
                            / vect.size();

    using namespace thrust::placeholders;
    thrust::transform(dVect.begin(), dVect.end(),
                      dVect.begin(), _1 /= norm);

    std::vector<double> retVect;
    thrust::copy(dVect.begin(), dVect.end(), retVect.begin());

    hipDeviceSynchronize();
    return retVect;
}

std::vector<double>
multiplyVec(const std::vector<double>& first,
            const std::vector<double>& second)
{
    thrust::device_vector<double> V1 = first;
    thrust::device_vector<double> V2 = second;
    thrust::device_vector<float> V3(first.size());

    thrust::transform(V1.begin(), V1.end(), V2.begin(), V3.begin(),
                      thrust::multiplies<double>());

    std::vector<double> retVect;
    thrust::copy(V3.begin(), V3.end(), retVect.begin());
    return retVect;
}

unsigned maxIndex(const std::vector<double>& vect)
{
    thrust::device_vector<double> d_vec = vect;
    thrust::device_vector<double>::iterator iter =
      thrust::max_element(d_vec.begin(), d_vec.end());
    unsigned position = iter - d_vec.begin();
//    float max_val = *iter;
    return position;
}

void gpuCalcFFT(const std::vector<double>& vectMeans,
                std::vector<double>& absVect)
{
//    std::cout << "daw: " << vectMeans.size();
    int n = vectMeans.size();
    double* in = new double[n];
    std::copy(vectMeans.begin(), vectMeans.end(), in);
    double* out = new double[2*n];
    // Allocate the buffer
    hipfftDoubleReal *d_in;
    hipfftDoubleComplex *d_out;
    unsigned int out_mem_size = sizeof(hipfftDoubleComplex)*n;
    unsigned int in_mem_size = sizeof(hipfftDoubleReal)*n;
    hipMalloc((void **)&d_in, in_mem_size);
    hipMalloc((void **)&d_out, out_mem_size);
    hipDeviceSynchronize();
    hipfftHandle plan;
    hipfftResult res = hipfftPlan1d(&plan, n, HIPFFT_D2Z, 1);
    if (res != HIPFFT_SUCCESS) {
        return;
    }
    hipMemcpy(d_in, in, in_mem_size, hipMemcpyHostToDevice);
    res = hipfftExecD2Z(plan, d_in, d_out);
    if (res != HIPFFT_SUCCESS) {
        return;
    }

    hipMemcpy(out, d_out, out_mem_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    std::vector<double> real, im;
    for(int i =1; i < 2*(n/2+1); i+=2) {
        real.push_back(out[i-1]);
        im.push_back(out[i]);
    }
//    obliczenie abs z tego
    for (int i =0; i< real.size(); ++i)
    {
        absVect.push_back(std::hypot(real.at(i), im.at(i)));
    }
}

}
